#include "hip/hip_runtime.h"
#include "ctranslate2/ops/dequantize.h"

#include "cuda/helpers.h"

namespace ctranslate2 {
  namespace ops {

    template <typename T>
    struct dequantize_func {
      __device__
      float operator()(float scale, T x) {
        return __fdividef(static_cast<float>(x), scale);
      }
    };

    template<>
    void Dequantize::dequantize<Device::CUDA, int8_t>(const StorageView& input,
                                                      const StorageView& scale,
                                                      StorageView& output) const {
      const dim_t depth = input.dim(-1);
      cuda::binary_transform(scale.data<float>(),
                             input.data<int8_t>(),
                             output.data<float>(),
                             input.size(),
                             dequantize_func<int8_t>(),
                             cuda::repeat_vec_depth<dim_t>(depth));
    }


    __device__ __forceinline__ float rescale(const int32_t c,
                                             const float a_scale,
                                             const float b_scale) {
      return __fdividef(__int2float_rn(c), a_scale * b_scale);
    }

    struct rescale_func {
      __device__
      float operator()(int32_t x, const thrust::tuple<float, float>& scales) {
        return rescale(x, thrust::get<0>(scales), thrust::get<1>(scales));
      }
    };

    struct rescale_and_add_bias_func {
      __device__
      float operator()(int32_t x, const thrust::tuple<float, float, float>& args) {
        return rescale(x, thrust::get<0>(args), thrust::get<1>(args)) + thrust::get<2>(args);
      }
    };

    template <bool transpose_a, bool transpose_b>
    static inline void dequantize_gemm_output_kernel(const int32_t* c,
                                                     const float* a_scales,
                                                     const float* b_scales,
                                                     const float* bias,
                                                     float* y,
                                                     dim_t batch_size,
                                                     dim_t depth) {
#define EXPAND(scales, transpose)                                       \
      thrust::make_permutation_iterator(                                \
        scales,                                                         \
        thrust::make_transform_iterator(                                \
          thrust::counting_iterator<int>(0),                            \
          typename std::conditional<                                    \
            transpose,                                                  \
            cuda::repeat_vec<int>,                                      \
            cuda::repeat_vec_depth<int>>::type(depth)))

      // y = c / (expand_dims(a_scales, trans_a ? 0 : 1) * expand_dims(b_scales, trans_b ? 0 : 1)
      // if bias: y += expand_dims(bias, 0)
      auto a_scales_it = EXPAND(a_scales, transpose_a);
      auto b_scales_it = EXPAND(b_scales, transpose_b);
      const dim_t size = batch_size * depth;
      if (bias) {
        auto args = thrust::make_zip_iterator(thrust::make_tuple(a_scales_it,
                                                                 b_scales_it,
                                                                 EXPAND(bias, true)));
        THRUST_CALL(thrust::transform,
                    c, c + size,
                    args,
                    y,
                    rescale_and_add_bias_func());
      } else {
        auto scales_it = thrust::make_zip_iterator(thrust::make_tuple(a_scales_it, b_scales_it));
        THRUST_CALL(thrust::transform,
                    c, c + size,
                    scales_it,
                    y,
                    rescale_func());
      }

#undef EXPAND
    }

    template<>
    void Dequantize::dequantize_gemm_output<Device::CUDA>(const StorageView& c,
                                                          const StorageView& a_scale,
                                                          const StorageView& b_scale,
                                                          const bool transpose_a,
                                                          const bool transpose_b,
                                                          const StorageView* bias,
                                                          StorageView& y) const {
      const dim_t batch_size = a_scale.size();
      const dim_t depth = c.dim(-1);

      const auto* c_data = c.data<int32_t>();
      const auto* a_scale_data = a_scale.data<float>();
      const auto* b_scale_data = b_scale.data<float>();
      const auto* bias_data = bias ? bias->data<float>() : nullptr;
      auto* y_data = y.data<float>();

      if (transpose_a && transpose_b)
        dequantize_gemm_output_kernel<true, true>(c_data, a_scale_data, b_scale_data, bias_data,
                                                  y_data, batch_size, depth);
      else if (transpose_a)
        dequantize_gemm_output_kernel<true, false>(c_data, a_scale_data, b_scale_data, bias_data,
                                                   y_data, batch_size, depth);
      else if (transpose_b)
        dequantize_gemm_output_kernel<false, true>(c_data, a_scale_data, b_scale_data, bias_data,
                                                   y_data, batch_size, depth);
      else
        dequantize_gemm_output_kernel<false, false>(c_data, a_scale_data, b_scale_data, bias_data,
                                                    y_data, batch_size, depth);
    }

  }
}
