#include "hip/hip_runtime.h"
#include "ctranslate2/ops/rotary.h"

#include "cuda/helpers.h"

namespace ctranslate2 {
  namespace ops {

    template <typename T>
    struct ComputeType {
      using type = T;
    };

#if !CUDA_CAN_USE_HALF
    template<>
    struct ComputeType<__half> {
      using type = float;
    };
#endif

#if !CUDA_CAN_USE_BF16_MATH
    template<>
    struct ComputeType<__hip_bfloat16> {
      using type = float;
    };
#endif

    template <typename T, bool interleave>
    __global__ void rotary_kernel(const T* x,
                                  const T* sin,
                                  const T* cos,
                                  T* y,
                                  const cuda::index_t max_time,
                                  const cuda::index_t head_size,
                                  const cuda::index_t ndims,
                                  const cuda::index_t depth,
                                  const bool transpose) {
      const auto time = transpose ? blockIdx.x % max_time : blockIdx.x / head_size;
      const auto middle = ndims / 2;

      x += blockIdx.x * depth;
      y += blockIdx.x * depth;

      sin += time * ndims;
      cos += time * ndims;

      using C = typename ComputeType<T>::type;

      for (cuda::index_t i = threadIdx.x; i < depth; i += blockDim.x) {
        if (i >= ndims)
          y[i] = x[i];
        else if (interleave)
          y[i] = C(x[i]) * C(cos[i]) + (i % 2 == 0 ? -C(x[i + 1]) : C(x[i - 1])) * C(sin[i]);
        else
          y[i] = C(x[i]) * C(cos[i]) + (i < middle ? -C(x[i + middle]) : C(x[i - middle])) * C(sin[i]);
      }
    }

    template <Device D, typename T>
    void Rotary::compute(const StorageView& input,
                         const StorageView& sin,
                         const StorageView& cos,
                         StorageView& output,
                         bool is_transposed) const {
      const dim_t max_time = is_transposed ? input.dim(-2) : input.dim(-3);
      const dim_t head_size = is_transposed ? input.dim(-3) : input.dim(-2);
      const dim_t depth = input.dim(-1);
      const dim_t ndims = _ndims == 0 ? depth : _ndims;

      const dim_t blocks = std::min(input.size() / depth, cuda::max_blocks);
      const dim_t threads = std::min(depth, cuda::max_threads);

      const auto* x = cuda::device_cast(input.data<T>());
      const auto* s = cuda::device_cast(sin.data<T>());
      const auto* c = cuda::device_cast(cos.data<T>());
      auto* y = cuda::device_cast(output.data<T>());

      using DeviceT = cuda::device_type<T>;

      if (_interleave)
        rotary_kernel<DeviceT, true><<<blocks, threads, 0, cuda::get_cuda_stream()>>>(
          x, s, c, y, max_time, head_size, ndims, depth, is_transposed);
      else
        rotary_kernel<DeviceT, false><<<blocks, threads, 0, cuda::get_cuda_stream()>>>(
          x, s, c, y, max_time, head_size, ndims, depth, is_transposed);
    }

#define DECLARE_IMPL(T)                                                 \
    template void                                                       \
    Rotary::compute<Device::CUDA, T>(const StorageView&,                \
                                     const StorageView&,                \
                                     const StorageView&,                \
                                     StorageView&,                       \
                                     bool) const;

    DECLARE_IMPL(float)
    DECLARE_IMPL(float16_t)
    DECLARE_IMPL(bfloat16_t)

  }
}
