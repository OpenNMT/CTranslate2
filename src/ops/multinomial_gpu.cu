#include "hip/hip_runtime.h"
#include "ctranslate2/ops/multinomial.h"

#include <chrono>
#include <memory>

#include <hipcub/hipcub.hpp>
#include <hiprand/hiprand_kernel.h>

#include "type_dispatch.h"
#include "cuda/helpers.h"

namespace ctranslate2 {
  namespace ops {

    template <typename hiprandState>
    __global__ void init_curand_states_kernel(hiprandState* states, unsigned long long seed) {
      const auto id = threadIdx.x + blockIdx.x * blockDim.x;
      hiprand_init(seed, id, 0, states + id);
    }

    template <typename hiprandState>
    class ScopedCurandStates {
    public:
      ScopedCurandStates(size_t num_states)
        : _allocator(get_allocator<Device::CUDA>())
        , _num_states(num_states)
        , _states(static_cast<hiprandState*>(_allocator.allocate(num_states * sizeof (hiprandState))))
      {
        constexpr size_t num_init_threads = 32;
        const size_t blocks = num_states / num_init_threads;
        init_curand_states_kernel<<<blocks, num_init_threads, 0, cuda::get_cuda_stream()>>>(
          _states, std::chrono::system_clock::now().time_since_epoch().count());
      }

      ~ScopedCurandStates() {
        _allocator.free(_states);
      }

      size_t num_states() const {
        return _num_states;
      }

      hiprandState* states() {
        return _states;
      }

    private:
      Allocator& _allocator;
      size_t _num_states;
      hiprandState* _states;
    };

    template <typename hiprandState>
    static hiprandState* get_curand_states(size_t num_states) {
      static thread_local std::unique_ptr<ScopedCurandStates<hiprandState>> states;
      if (!states || num_states > states->num_states())
        states = std::make_unique<ScopedCurandStates<hiprandState>>(num_states);
      return states->states();
    }

    // Structure tracking the prefix sum of the previous block of threads.
    template <typename T>
    struct BlockPrefixSum {
      T prefix_sum = 0;

      __device__ T operator()(T block_aggregate) {
        T old_prefix = prefix_sum;
        prefix_sum += block_aggregate;
        return old_prefix;
      }
    };

    constexpr dim_t num_threads = 256;

    template <typename In, typename Out>
    __global__ void multinomial_kernel(const In* probs,
                                       cuda::index_t class_size,
                                       Out* output,
                                       hiprandStatePhilox4_32_10_t* states) {
      __shared__ float random_sample;
      if (threadIdx.x == 0)
        random_sample = hiprand_uniform(states + blockIdx.x);
      __syncthreads();

      typedef hipcub::BlockScan<float, num_threads> BlockScan;
      __shared__ typename BlockScan::TempStorage presum_temp_storage;

      BlockPrefixSum<float> prefix_op;
      Out candidate = class_size - 1;

      // In this loop we ensure that all threads do the same work,
      // even if some thread IDs are out of bounds.
      for (cuda::index_t offset = 0; offset < class_size; offset += blockDim.x) {
        const auto i = offset + threadIdx.x;
        float prob = i < class_size ? float(probs[blockIdx.x * class_size + i]) : 0.f;
        float prefix_sum_prob;
        BlockScan(presum_temp_storage).InclusiveSum(prob, prefix_sum_prob, prefix_op);
        __syncthreads();

        if (i < candidate && prefix_sum_prob >= random_sample)
          candidate = i;
      }

      // Get the first candidate.
      typedef hipcub::BlockReduce<Out, num_threads> BlockReduce;
      __shared__ typename BlockReduce::TempStorage min_temp_storage;
      Out first_candidate = BlockReduce(min_temp_storage).Reduce(candidate,
                                                                 cuda::minimum<Out>(),
                                                                 class_size);

      if (threadIdx.x == 0)
        output[blockIdx.x] = first_candidate;
    }

    template <Device D, typename T>
    void Multinomial::compute(const StorageView& input, StorageView& output) const {
      if (_sample_size != 1) {
        // The current CUDA kernel only returns a single sample per batch, so fallback on CPU.
        StorageView output_host(output.shape(), output.dtype());
        dispatch(input.to(Device::CPU), output_host);
        output.copy_from(output_host);
        return;
      }

      const dim_t depth = input.dim(-1);
      const dim_t batch_size = input.size() / depth;
      const dim_t blocks = std::min(batch_size, cuda::max_blocks);

      // Get one hiprand state per block.
      auto* curand_states = get_curand_states<hiprandStatePhilox4_32_10_t>(blocks);

      multinomial_kernel<<<blocks, num_threads, 0, cuda::get_cuda_stream()>>>(
        cuda::device_cast(input.data<T>()),
        depth,
        output.data<int32_t>(),
        curand_states);
    }

#define DECLARE_IMPL(T)                                                 \
    template void                                                       \
    Multinomial::compute<Device::CUDA, T>(const StorageView& input,     \
                                          StorageView& output) const;

    DECLARE_IMPL(float)
    DECLARE_IMPL(float16_t)

  }
}
