#include "hip/hip_runtime.h"
#include "ctranslate2/ops/mean.h"

#include "type_dispatch.h"
#include "cuda/helpers.h"

namespace ctranslate2 {
  namespace ops {

    template <typename T, typename AccumT>
    __global__ void mean_kernel(const T* input,
                                const cuda::index_t outer_size,
                                const cuda::index_t axis_size,
                                const cuda::index_t inner_size,
                                T* output) {
      typedef hipcub::BlockReduce<AccumT, 256> BlockReduce;
      __shared__ typename BlockReduce::TempStorage temp_storage;

      const cuda::index_t i = blockIdx.x / inner_size;
      const cuda::index_t j = blockIdx.x % inner_size;

      AccumT block_sum = 0;
      for (cuda::index_t k = threadIdx.x; k < axis_size; k += blockDim.x) {
        block_sum += AccumT(input[i * axis_size * inner_size + k * inner_size + j]);
      }

      AccumT sum = BlockReduce(temp_storage).Sum(block_sum);

      if (threadIdx.x == 0) {
        output[blockIdx.x] = sum / AccumT(axis_size);
      }
    }

    template <Device D, typename T>
    void Mean::compute(const StorageView& input,
                       const dim_t outer_size,
                       const dim_t axis_size,
                       const dim_t inner_size,
                       StorageView& output) const {
      const dim_t blocks = std::min(outer_size * inner_size, cuda::max_blocks);
      const dim_t threads = 256;
      mean_kernel<cuda::device_type<T>, float><<<blocks, threads, 0, cuda::get_cuda_stream()>>>(
        cuda::device_cast(input.data<T>()),
        outer_size,
        axis_size,
        inner_size,
        cuda::device_cast(output.data<T>()));
    }

#define DECLARE_IMPL(T)                                         \
    template void                                               \
    Mean::compute<Device::CUDA, T>(const StorageView& input,    \
                                   const dim_t outer_size,      \
                                   const dim_t axis_size,       \
                                   const dim_t inner_size,      \
                                   StorageView& output) const;

    DECLARE_IMPL(float)
    DECLARE_IMPL(float16_t)

  }
}
