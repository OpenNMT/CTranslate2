#include "hip/hip_runtime.h"
#include "ctranslate2/ops/bias_add.h"

#include "type_dispatch.h"
#include "cuda/helpers.h"

namespace ctranslate2 {
  namespace ops {

    template <typename T, typename AddFunc, typename Epilogue>
    __global__ void bias_add_kernel(const T* value,
                                    const T* bias,
                                    T* output,
                                    cuda::index_t depth,
                                    const AddFunc& add_func,
                                    const Epilogue& epilogue) {
      const cuda::index_t i = blockIdx.x;
      for (cuda::index_t j = threadIdx.x; j < depth; j += blockDim.x) {
        const cuda::index_t index = i * depth + j;
        output[index] = epilogue(add_func(value[index], bias[j]));
      }
    }

    template <Device D, typename T>
    void BiasAdd::compute(const StorageView& value,
                          const StorageView& bias,
                          StorageView& output) const {
      const dim_t depth = bias.size();
      const dim_t batch_size = value.size() / depth;
      const dim_t blocks = std::min(batch_size, cuda::max_blocks);
      const dim_t threads = std::min(depth, cuda::max_threads);

      using DeviceT = cuda::device_type<T>;
      const auto* x = cuda::device_cast(value.data<T>());
      const auto* b = cuda::device_cast(bias.data<T>());
      auto* y = cuda::device_cast(output.data<T>());

      if (!_activation_type) {
        bias_add_kernel<<<blocks, threads, 0, cuda::get_cuda_stream()>>>(
          x, b, y, depth, cuda::plus<DeviceT>(), thrust::identity<DeviceT>());

      } else {
        switch (*_activation_type) {

        case ActivationType::ReLU:
          bias_add_kernel<<<blocks, threads, 0, cuda::get_cuda_stream()>>>(
            x, b, y, depth, cuda::plus<DeviceT>(), cuda::relu_func<DeviceT>());
          break;

        case ActivationType::GELU:
          bias_add_kernel<<<blocks, threads, 0, cuda::get_cuda_stream()>>>(
            x, b, y, depth, cuda::plus<DeviceT>(), cuda::gelu_func<DeviceT>());
          break;

        case ActivationType::GELUTanh:
          bias_add_kernel<<<blocks, threads, 0, cuda::get_cuda_stream()>>>(
            x, b, y, depth, cuda::plus<DeviceT>(), cuda::gelu_tanh_func<DeviceT>());
          break;

        case ActivationType::GELUSigmoid:
          bias_add_kernel<<<blocks, threads, 0, cuda::get_cuda_stream()>>>(
            x, b, y, depth, cuda::plus<DeviceT>(), cuda::gelu_sigmoid_func<DeviceT>());
          break;

        case ActivationType::Swish:
          bias_add_kernel<<<blocks, threads, 0, cuda::get_cuda_stream()>>>(
            x, b, y, depth, cuda::plus<DeviceT>(), cuda::swish_func<DeviceT>());
          break;

        case ActivationType::Tanh:
          bias_add_kernel<<<blocks, threads, 0, cuda::get_cuda_stream()>>>(
            x, b, y, depth, cuda::plus<DeviceT>(), cuda::tanh_func<DeviceT>());
          break;
        }
      }
    }

#define DECLARE_IMPL(T)                                                 \
    template void                                                       \
    BiasAdd::compute<Device::CUDA, T>(const StorageView& value,         \
                                      const StorageView& bias,          \
                                      StorageView& output) const;

    DECLARE_IMPL(float)
    DECLARE_IMPL(float16_t)
    DECLARE_IMPL(bfloat16_t)

  }
}
