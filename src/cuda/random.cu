#include "hip/hip_runtime.h"
#include "random.h"

#include <memory>

#include "ctranslate2/allocator.h"
#include "ctranslate2/random.h"
#include "ctranslate2/utils.h"
#include "utils.h"

namespace ctranslate2 {
  namespace cuda {

    template <typename hiprandState>
    __global__ void init_curand_states_kernel(hiprandState* states, unsigned long long seed) {
      const auto id = threadIdx.x + blockIdx.x * blockDim.x;
      hiprand_init(seed, id, 0, states + id);
    }

    template <typename hiprandState>
    class ScopedCurandStates {
    public:
      ScopedCurandStates(size_t num_states)
        : _allocator(get_allocator<Device::CUDA>())
      {
        constexpr size_t num_init_threads = 32;
        const size_t blocks = ceil_divide(num_states, num_init_threads);
        _num_states = blocks * num_init_threads;
        _states = static_cast<hiprandState*>(_allocator.allocate(_num_states * sizeof (hiprandState)));
        init_curand_states_kernel<<<blocks, num_init_threads, 0, cuda::get_cuda_stream()>>>(
          _states, get_random_seed());
      }

      ~ScopedCurandStates() {
        _allocator.free(_states);
      }

      size_t num_states() const {
        return _num_states;
      }

      hiprandState* states() {
        return _states;
      }

    private:
      Allocator& _allocator;
      size_t _num_states;
      hiprandState* _states;
    };

    hiprandStatePhilox4_32_10_t* get_curand_states(size_t num_states) {
      static thread_local std::unique_ptr<ScopedCurandStates<hiprandStatePhilox4_32_10_t>> states;
      if (!states || num_states > states->num_states())
        states = std::make_unique<ScopedCurandStates<hiprandStatePhilox4_32_10_t>>(num_states);
      return states->states();
    }

  }
}
