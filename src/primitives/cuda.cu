#include "hip/hip_runtime.h"
#include "ctranslate2/primitives/primitives.h"

#include <cmath>
#include <type_traits>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <cub/util_allocator.cuh>

#include "../cuda/utils.h"

namespace ctranslate2 {

  template <typename T1, typename T2, typename UnaryFunction>
  void unary_transform(const T1* x, T2* y, dim_t size, const UnaryFunction& op) {
    THRUST_CALL(thrust::transform, x, x + size, y, op);
  }

  template <typename T1, typename T2, typename T3, typename BinaryFunction>
  void binary_transform(const T1* a, const T2* b, T3* c, dim_t size, const BinaryFunction& op) {
    THRUST_CALL(thrust::transform, a, a + size, b, c, op);
  }

  template <typename T1, typename T2, typename T3, typename BinaryFunction, typename IndexFunction>
  void binary_transform(T1 a, T2 b, T3 c, dim_t size,
                        const BinaryFunction& op, const IndexFunction& index_a) {
    auto index_it = thrust::make_transform_iterator(thrust::counting_iterator<dim_t>(0), index_a);
    auto a_it = thrust::make_permutation_iterator(a, index_it);
    THRUST_CALL(thrust::transform, a_it, a_it + size, b, c, op);
  }

  // perm_fun is a functor that takes the index in the permuted iterator and
  // return the index in the original iterator.
  template <typename T, typename PermFunction>
  void permute(const T* x, T* y, dim_t size, const PermFunction& perm_fun) {
    auto ind_it = thrust::counting_iterator<dim_t>(0);
    auto perm_ind_it = thrust::make_transform_iterator(ind_it, perm_fun);
    auto perm_it = thrust::make_permutation_iterator(x, perm_ind_it);
    THRUST_CALL(thrust::copy_n, perm_it, size, y);
  }

  template <typename T>
  struct repeat_vec : thrust::unary_function<T, T> {
    T _size;
    repeat_vec(T size)
      : _size(size) {
    }
    __host__ __device__
    T operator()(const T i) {
      return i % _size;
    }
  };

  template <typename T>
  struct repeat_vec_depth : thrust::unary_function<T, T> {
    T _size;
    repeat_vec_depth(T size)
      : _size(size) {
    }
    __host__ __device__
    T operator()(const T i) {
      return i / _size;
    }
  };


  static const cuda::CachingAllocatorConfig allocator_config = cuda::get_caching_allocator_config();
  static hipcub::CachingDeviceAllocator allocator(
    allocator_config.bin_growth,
    allocator_config.min_bin,
    allocator_config.max_bin,
    allocator_config.max_cached_bytes);

  template<>
  void primitives<Device::CUDA>::set_device(int index) {
    CUDA_CHECK(hipSetDevice(index));
  }

  template<>
  int primitives<Device::CUDA>::get_device() {
    int index;
    CUDA_CHECK(hipGetDevice(&index));
    return index;
  }

  template<>
  void* primitives<Device::CUDA>::alloc_data(dim_t size, int device_index) {
    if (device_index < 0)
      device_index = hipcub::CachingDeviceAllocator::INVALID_DEVICE_ORDINAL;
    void* data = nullptr;
    CUDA_CHECK(allocator.DeviceAllocate(device_index, &data, size, cuda::get_cuda_stream()));
    return data;
  }

  template<>
  void primitives<Device::CUDA>::free_data(void* data, int device_index) {
    CUDA_CHECK(allocator.DeviceFree(device_index, data));
  }

  template<>
  void primitives<Device::CUDA>::clear_cache() {
    CUDA_CHECK(allocator.FreeAllCached());
  }

  template<>
  template <typename T>
  T primitives<Device::CUDA>::deref(const T* x, dim_t index) {
    T val = T();
    cross_device_primitives<Device::CUDA, Device::CPU>::copy(x + index, &val, 1);
    return val;
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::fill(T* x, T a, dim_t size) {
    THRUST_CALL(thrust::fill_n, x, size, a);
  }
  template<>
  template <typename T>
  void primitives<Device::CUDA>::strided_fill(T* x, T a, dim_t inc_x, dim_t size) {
    auto it = thrust::make_permutation_iterator(
      x, thrust::make_transform_iterator(thrust::counting_iterator<dim_t>(0),
                                         thrust::placeholders::_1 * inc_x));
    THRUST_CALL(thrust::fill_n, it, size, a);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::copy(const T* x, T* y, dim_t size) {
    CUDA_CHECK(hipMemcpyAsync(y, x, size * sizeof (T),
                               hipMemcpyDeviceToDevice, cuda::get_cuda_stream()));
  }

  template<>
  template <typename T>
  T primitives<Device::CUDA>::sum(const T* array, dim_t size) {
    return THRUST_CALL(thrust::reduce, array, array + size);
  }

  template<>
  template <typename T>
  dim_t primitives<Device::CUDA>::max_element(const T* array, dim_t size) {
    const auto* max = THRUST_CALL(thrust::max_element, array, array + size);
    return static_cast<dim_t>(max - array);
  }

  template<>
  template <typename T>
  T primitives<Device::CUDA>::max(const T* array, dim_t size) {
    const auto* max = THRUST_CALL(thrust::max_element, array, array + size);
    return deref(max, 0);
  }

  template <typename T1, typename T2>
  struct greater_tuple {
    __device__ __host__
    thrust::tuple<T1, T2> operator()(const thrust::tuple<T1, T2>& a,
                                     const thrust::tuple<T1, T2>& b) const {
      if (a > b)
        return a;
      else
        return b;
    }
  };

  template<>
  template <typename T>
  void primitives<Device::CUDA>::row_max(const T* x,
                                         const dim_t rows,
                                         const dim_t cols,
                                         T* values,
                                         int32_t* indices) {
    auto keys_it = thrust::make_transform_iterator(thrust::counting_iterator<int32_t>(0),
                                                   repeat_vec_depth<int32_t>(cols));
    auto ids_it = thrust::make_transform_iterator(thrust::counting_iterator<int32_t>(0),
                                                  repeat_vec<int32_t>(cols));

    THRUST_CALL(thrust::reduce_by_key,
                keys_it, keys_it + (rows * cols),
                thrust::make_zip_iterator(thrust::make_tuple(x, ids_it)),
                thrust::make_discard_iterator(),
                thrust::make_zip_iterator(thrust::make_tuple(values, indices)),
                thrust::equal_to<int32_t>(),
                greater_tuple<T, int32_t>());
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::add(T a, const T* x, T* y, dim_t size) {
    unary_transform(x, y, size, thrust::placeholders::_1 + a);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::add(const T* a, const T* b, T* c, dim_t size) {
    binary_transform(a, b, c, size, thrust::plus<T>());
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::add_batch_broadcast(const T* a, const T* b, T* c,
                                                     dim_t a_size, dim_t b_size) {
    binary_transform(a, b, c, b_size, thrust::plus<T>(), repeat_vec<dim_t>(a_size));
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::add_depth_broadcast(const T* a, const T* b, T* c,
                                                     dim_t a_size, dim_t b_size) {
    binary_transform(a, b, c, b_size, thrust::plus<T>(), repeat_vec_depth<dim_t>(b_size / a_size));
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::sub(const T* a, const T* b, T* c, dim_t size) {
    binary_transform(a, b, c, size, thrust::minus<T>());
  }

  template<typename T>
  struct min_func : public thrust::unary_function<T, T> {
    T a_;
    __host__ __device__
    min_func(T a):a_(a){}
    __host__ __device__
    T operator()(T x) {return x > a_ ? a_ : x;}
  };

  template<>
  template <typename T>
  void primitives<Device::CUDA>::min(T a, const T* x, T* y, dim_t size) {
    unary_transform(x, y, size, min_func<T>(a));
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::min(const T* a, const T* b, T* c, dim_t size) {
    binary_transform(a, b, c, size, thrust::minimum<T>());
  }

  template<typename T>
  struct max_func : public thrust::unary_function<T, T> {
    T a_;
    __host__ __device__
    max_func(T a):a_(a){}
    __host__ __device__
    T operator()(T x) {return x > a_ ? x : a_;}
  };

  template<>
  template <typename T>
  void primitives<Device::CUDA>::max(T a, const T* x, T* y, dim_t size) {
    unary_transform(x, y, size, max_func<T>(a));
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::max(const T* a, const T* b, T* c, dim_t size) {
    binary_transform(a, b, c, size, thrust::maximum<T>());
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::mul(T a, const T* x, T* y, dim_t size) {
    unary_transform(x, y, size, thrust::placeholders::_1 * a);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::mul(const T* a, const T* b, T* c, dim_t size) {
    binary_transform(a, b, c, size, thrust::multiplies<T>());
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::mul_batch_broadcast(const T* a, const T* b, T* c,
                                                     dim_t a_size, dim_t b_size) {
    binary_transform(a, b, c, b_size, thrust::multiplies<T>(), repeat_vec<dim_t>(a_size));
  }

  struct absolute_maximum_func : public thrust::binary_function<float, float, float> {
    __host__ __device__
    float operator()(float a, float b) {
      return fmaxf(fabsf(a), fabsf(b));
    }
  };

  template <typename T>
  class quantize_func : public thrust::binary_function<float, float, T> {
  private:
    float _shift;
  public:
    quantize_func(float shift)
      : _shift(shift) {
    }
    __host__ __device__
    T operator()(float scale, float x) {
      return static_cast<T>(x * scale + _shift);
    }
  };

  template<>
  void primitives<Device::CUDA>::quantize_batch(const float* x,
                                                float* scales,
                                                int8_t* qx,
                                                dim_t batch_size,
                                                dim_t depth,
                                                float shift) {
    const dim_t size = batch_size * depth;

    // Assign 1 key per batch.
    auto keys_it = thrust::make_transform_iterator(thrust::counting_iterator<int>(0),
                                                   repeat_vec_depth<int>(depth));

    // scales = 127.0 / reduce_max(abs(x), axis=1)
    THRUST_CALL(thrust::reduce_by_key,
                keys_it, keys_it + size,
                x,
                thrust::make_discard_iterator(),
                thrust::make_transform_output_iterator(
                  scales, static_cast<float>(127) / thrust::placeholders::_1),
                thrust::equal_to<int>(),
                absolute_maximum_func());

    // qx = x * expand_dims(scales, 1)
    binary_transform(scales, x, qx, size,
                     quantize_func<int8_t>(shift),
                     repeat_vec_depth<dim_t>(depth));
  }

  template <typename T>
  class dequantize_func : public thrust::binary_function<float, T, float> {
  private:
    float _shift;
  public:
    dequantize_func(float shift)
      : _shift(shift) {
    }
    __device__
    float operator()(float scale, T x) {
      return __fdividef(static_cast<float>(x) - _shift, scale);
    }
  };

  template<>
  template<>
  void primitives<Device::CUDA>::dequantize_batch(const int8_t* x, const float* scale, float* y,
                                                  dim_t x_size, dim_t scale_size, float shift) {
    binary_transform(scale, x, y, x_size,
                     dequantize_func<int8_t>(shift),
                     repeat_vec_depth<dim_t>(x_size / scale_size));
  }

  struct rescale_func : public thrust::binary_function<int32_t, thrust::tuple<float, float>, float> {
    __device__
    float operator()(int32_t x, const thrust::tuple<float, float>& scales) {
      return __fdividef(__int2float_rn(x), (thrust::get<0>(scales) * thrust::get<1>(scales)));
    }
  };

  template <bool transpose_a, bool transpose_b>
  static void rescale_output_impl(const int32_t* c,
                                  const float* a_scales,
                                  const float* b_scales,
                                  float* y,
                                  dim_t batch_size,
                                  dim_t depth) {
#define EXPAND(scales, transpose)                                       \
    thrust::make_permutation_iterator(                                  \
      scales,                                                           \
      thrust::make_transform_iterator(                                  \
        thrust::counting_iterator<int>(0),                              \
        typename std::conditional<transpose, repeat_vec<int>, repeat_vec_depth<int>>::type(depth)))

    // y = c / (expand_dims(a_scales, trans_a ? 0 : 1) * expand_dims(b_scales, trans_b ? 0 : 1)
    auto a_scales_it = EXPAND(a_scales, transpose_a);
    auto b_scales_it = EXPAND(b_scales, transpose_b);
    auto scales_it = thrust::make_zip_iterator(thrust::make_tuple(a_scales_it, b_scales_it));
    const dim_t size = batch_size * depth;
    THRUST_CALL(thrust::transform,
                c, c + size,
                scales_it,
                y,
                rescale_func());

#undef EXPAND
  }

  template<>
  void primitives<Device::CUDA>::rescale_output(const int32_t* c,
                                                const float* a_scales,
                                                const float* b_scales,
                                                const bool transpose_a,
                                                const bool transpose_b,
                                                float* y,
                                                dim_t batch_size,
                                                dim_t depth) {
    if (transpose_a && transpose_b)
      rescale_output_impl<true, true>(c, a_scales, b_scales, y, batch_size, depth);
    else if (transpose_a)
      rescale_output_impl<true, false>(c, a_scales, b_scales, y, batch_size, depth);
    else if (transpose_b)
      rescale_output_impl<false, true>(c, a_scales, b_scales, y, batch_size, depth);
    else
      rescale_output_impl<false, false>(c, a_scales, b_scales, y, batch_size, depth);
  }

  struct relu_func : public thrust::unary_function<float, float> {
    __host__ __device__
    float operator()(float x) { return fmaxf(x, 0); }
  };

  template<>
  void primitives<Device::CUDA>::relu(const float* x, float* y, dim_t size) {
    unary_transform(x, y, size, relu_func());
  }

  struct gelu_func : public thrust::unary_function<float, float> {
    float _scale;
    gelu_func(float scale)
      : _scale(scale) {
    }
    __host__ __device__
    float operator()(float x) {
      return 0.5f * x * (1.f + tanhf(_scale * (x + 0.044715f * powf(x, 3.f))));
    }
  };

  template<>
  void primitives<Device::CUDA>::gelu(const float* x, float* y, dim_t size) {
    static const float pi = std::acos(-1.f);
    static const float scale = std::sqrt(2.f / pi);
    unary_transform(x, y, size, gelu_func(scale));
  }

  template <typename T>
  struct perm_indices_2d : public thrust::unary_function<T, T> {
    T _rows, _cols;
    perm_indices_2d(T rows, T cols)
      : _rows(rows)
      , _cols(cols) {
    }
    __host__ __device__
    T operator()(const T i) const {
      const T i0 = i / _rows;
      const T i1 = i % _rows;
      return i1 * _cols + i0;
    }
  };

  template<>
  template <typename T>
  void primitives<Device::CUDA>::transpose_2d(const T* a, const dim_t* dims, T* b) {
    permute(a, b, dims[0] * dims[1], perm_indices_2d<dim_t>(dims[0], dims[1]));
  }

  template <typename T>
  struct perm_indices_3d : public thrust::unary_function<T, T> {
    T _a_ps0, _a_ps1, _a_ps2; // Permuted strides of the original array.
    T _b_d0, _b_d1, _b_d2;    // Dimension of the permutated array.
    T _b_s0, _b_s1, _b_s2;    // Strides of the permutated array.
    perm_indices_3d(const T* dims, const T* perm) {
      const T a_stride[3] = {dims[1] * dims[2], dims[2], 1};
      _a_ps0 = a_stride[perm[0]];
      _a_ps1 = a_stride[perm[1]];
      _a_ps2 = a_stride[perm[2]];
      _b_d0 = dims[perm[0]];
      _b_d1 = dims[perm[1]];
      _b_d2 = dims[perm[2]];
      _b_s0 = _b_d1 * _b_d2;
      _b_s1 = _b_d2;
      _b_s2 = 1;
    }
    __host__ __device__
    T operator()(const T i) const {
      const T i0 = i / _b_s0;
      const T i1 = i / _b_s1 % _b_d1;
      const T i2 = i % _b_d2;
      return i0 * _a_ps0 + i1 * _a_ps1 + i2 * _a_ps2;
    }
  };

  template<>
  template <typename T>
  void primitives<Device::CUDA>::transpose_3d(const T* a,
                                              const dim_t* dims,
                                              const dim_t* perm,
                                              T* b) {
    permute(a, b, dims[0] * dims[1] * dims[2], perm_indices_3d<dim_t>(dims, perm));
  }

  template <typename T>
  struct perm_indices_4d : public thrust::unary_function<T, T> {
    T _a_ps0, _a_ps1, _a_ps2, _a_ps3; // Permuted strides of the original array.
    T _b_d0, _b_d1, _b_d2, _b_d3;    // Dimension of the permutated array.
    T _b_s0, _b_s1, _b_s2, _b_s3;    // Strides of the permutated array.
    perm_indices_4d(const T* dims, const T* perm) {
      const T a_stride[4] = {dims[1] * dims[2] * dims[3], dims[2] * dims[3], dims[3], 1};
      _a_ps0 = a_stride[perm[0]];
      _a_ps1 = a_stride[perm[1]];
      _a_ps2 = a_stride[perm[2]];
      _a_ps3 = a_stride[perm[3]];
      _b_d0 = dims[perm[0]];
      _b_d1 = dims[perm[1]];
      _b_d2 = dims[perm[2]];
      _b_d3 = dims[perm[3]];
      _b_s0 = _b_d1 * _b_d2 * _b_d3;
      _b_s1 = _b_d2 * _b_d3;
      _b_s2 = _b_d3;
      _b_s3 = 1;
    }
    __host__ __device__
    T operator()(const T i) const {
      const T i0 = i / _b_s0;
      const T i1 = i / _b_s1 % _b_d1;
      const T i2 = i / _b_s2 % _b_d2;
      const T i3 = i % _b_d3;
      return i0 * _a_ps0 + i1 * _a_ps1 + i2 * _a_ps2 + i3 * _a_ps3;
    }
  };

  template<>
  template <typename T>
  void primitives<Device::CUDA>::transpose_4d(const T* a,
                                              const dim_t* dims,
                                              const dim_t* perm,
                                              T* b) {
    permute(a, b, dims[0] * dims[1] * dims[2] * dims[3], perm_indices_4d<dim_t>(dims, perm));
  }

  template<>
  template<>
  void primitives<Device::CUDA>::gemm(const float* a, const float* b,
                                      bool, bool,
                                      bool transpose_a, bool transpose_b,
                                      dim_t m, dim_t n, dim_t k,
                                      float alpha, float beta,
                                      float* c,
                                      const float*) {
    // Memo: cuBLAS assumes column-major storage.

    const int lda = transpose_a ? m : k;
    const int ldb = transpose_b ? k : n;
    const int ldc = n;

    const hipblasOperation_t transa = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transb = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    CUBLAS_CHECK(hipblasSgemm(cuda::get_cublas_handle(),
                             transb, transa,
                             n, m, k,
                             &alpha,
                             b, ldb,
                             a, lda,
                             &beta,
                             c, ldc));
  }

  template<>
  template<>
  void primitives<Device::CUDA>::gemm(const int8_t* a, const int8_t* b,
                                      bool, bool,
                                      bool transpose_a, bool transpose_b,
                                      dim_t m, dim_t n, dim_t k,
                                      float alpha, float beta,
                                      int32_t* c,
                                      const int32_t*) {
    const int lda = transpose_a ? m : k;
    const int ldb = transpose_b ? k : n;
    const int ldc = n;

    const hipblasOperation_t transa = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transb = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    int32_t alpha_i = alpha;
    int32_t beta_i = beta;

    // cuBLAS assumes column-major storage, so swap a and b accordingly.
    CUBLAS_CHECK(hipblasGemmEx(cuda::get_cublas_handle(),
                              transb, transa,
                              n, m, k,
                              &alpha_i,
                              b, HIP_R_8I, ldb,
                              a, HIP_R_8I, lda,
                              &beta_i,
                              c, HIP_R_32I, ldc,
                              HIP_R_32I,
                              CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }

  template<>
  template<>
  void primitives<Device::CUDA>::gemm_batch(const float* a, const float* b,
                                            bool transpose_a, bool transpose_b,
                                            dim_t batch_size,
                                            dim_t m, dim_t n, dim_t k,
                                            float alpha, float beta,
                                            float* c) {
    // Memo: cuBLAS assumes column-major storage.

    const int lda = transpose_a ? m : k;
    const int ldb = transpose_b ? k : n;
    const int ldc = n;

    const long long int stridea = m * k;
    const long long int strideb = k * n;
    const long long int stridec = m * n;

    const hipblasOperation_t transa = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transb = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    CUBLAS_CHECK(hipblasSgemmStridedBatched(cuda::get_cublas_handle(),
                                           transb, transa,
                                           n, m, k,
                                           &alpha,
                                           b, ldb, strideb,
                                           a, lda, stridea,
                                           &beta,
                                           c, ldc, stridec,
                                           batch_size));
  }

  struct exp_func : public thrust::unary_function<float, float> {
    __host__ __device__
    float operator()(float x) { return expf(x); }
  };

  template<>
  void primitives<Device::CUDA>::exp(const float* x, float* y, dim_t size) {
    unary_transform(x, y, size, exp_func());
  }

  struct log_func : public thrust::unary_function<float, float> {
    __host__ __device__
    float operator()(float x) { return logf(x); }
  };

  template<>
  void primitives<Device::CUDA>::log(const float* x, float* y, dim_t size) {
    unary_transform(x, y, size, log_func());
  }


  template<>
  template <typename T>
  void cross_device_primitives<Device::CPU, Device::CUDA>::copy(const T* x, T* y, dim_t size) {
    CUDA_CHECK(hipMemcpyAsync(y, x, size * sizeof (T), hipMemcpyHostToDevice, cuda::get_cuda_stream()));
  }

  template<>
  template <typename T>
  void cross_device_primitives<Device::CUDA, Device::CPU>::copy(const T* x, T* y, dim_t size) {
    CUDA_CHECK(hipMemcpyAsync(y, x, size * sizeof (T), hipMemcpyDeviceToHost, cuda::get_cuda_stream()));
  }

#define DECLARE_IMPL(T)                                                 \
  template T                                                            \
  primitives<Device::CUDA>::deref(const T* x, dim_t index);             \
  template void                                                         \
  primitives<Device::CUDA>::fill(T* x, T a, dim_t size);                \
  template void                                                         \
  primitives<Device::CUDA>::strided_fill(T* x, T a, dim_t inc_x, dim_t size); \
  template void                                                         \
  primitives<Device::CUDA>::copy<T>(const T* x, T* y, dim_t size);      \
  template T                                                            \
  primitives<Device::CUDA>::sum(const T* array, dim_t size);            \
  template dim_t                                                        \
  primitives<Device::CUDA>::max_element(const T* array, dim_t size);    \
  template T                                                            \
  primitives<Device::CUDA>::max(const T* array, dim_t size);            \
  template void                                                         \
  primitives<Device::CUDA>::row_max(const T* x,                         \
                                    const dim_t rows,                   \
                                    const dim_t cols,                   \
                                    T* values,                          \
                                    int32_t* indices);                  \
  template void                                                         \
  primitives<Device::CUDA>::add(T a, const T* x, T* y, dim_t size);     \
  template void                                                         \
  primitives<Device::CUDA>::add(const T* a, const T* b, T* c, dim_t size); \
  template void                                                         \
  primitives<Device::CUDA>::add_batch_broadcast(const T* a, const T* b, \
                                                T* c, dim_t a_size, dim_t b_size); \
  template void                                                         \
  primitives<Device::CUDA>::add_depth_broadcast(const T* a, const T* b, \
                                                T* c, dim_t a_size, dim_t b_size); \
  template void                                                         \
  primitives<Device::CUDA>::sub(const T* a, const T* b, T* c, dim_t size); \
  template void                                                         \
  primitives<Device::CUDA>::min(T a, const T* x, T* y, dim_t size);      \
  template void                                                         \
  primitives<Device::CUDA>::min(const T* a, const T* b, T* c, dim_t size); \
  template void                                                         \
  primitives<Device::CUDA>::max(T a, const T* x, T* y, dim_t size);     \
  template void                                                         \
  primitives<Device::CUDA>::max(const T* a, const T* b, T* c, dim_t size); \
  template void                                                         \
  primitives<Device::CUDA>::mul(T a, const T* x, T* y, dim_t size);     \
  template void                                                         \
  primitives<Device::CUDA>::mul(const T* a, const T* b, T* c, dim_t size); \
  template void                                                         \
  primitives<Device::CUDA>::mul_batch_broadcast(const T* a, const T* b, \
                                                T* c, dim_t a_size, dim_t b_size); \
  template void                                                         \
  primitives<Device::CUDA>::transpose_2d(const T* a,                    \
                                         const dim_t* dims,             \
                                         T* b);                         \
  template void                                                         \
  primitives<Device::CUDA>::transpose_3d(const T* a,                    \
                                         const dim_t* dims,             \
                                         const dim_t* perm,             \
                                         T* b);                         \
  template void                                                         \
  primitives<Device::CUDA>::transpose_4d(const T* a,                    \
                                         const dim_t* dims,             \
                                         const dim_t* perm,             \
                                         T* b);                         \
  template void                                                         \
  cross_device_primitives<Device::CPU, Device::CUDA>::copy<T>(const T*, T*, dim_t); \
  template void                                                         \
  cross_device_primitives<Device::CUDA, Device::CPU>::copy<T>(const T*, T*, dim_t);

  DECLARE_ALL_TYPES(DECLARE_IMPL)

}

