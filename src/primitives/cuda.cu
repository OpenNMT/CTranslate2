#include "hip/hip_runtime.h"
#include "ctranslate2/primitives/primitives.h"

#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <cub/util_allocator.cuh>

#include "../cuda/utils.h"

namespace ctranslate2 {

  template <typename T, typename UnaryFunction>
  void unary_transform(const T* x, T* y, dim_t size, UnaryFunction op) {
    THRUST_CALL(thrust::transform, x, x + size, y, op);
  }

  template <typename T, typename BinaryFunction>
  void binary_transform(const T* a, const T* b, T* c, dim_t size, BinaryFunction op) {
    THRUST_CALL(thrust::transform, a, a + size, b, c, op);
  }

  template <typename T1, typename T2, typename T3, typename BinaryFunction, typename IndexFunction>
  void binary_transform(T1 a, T2 b, T3 c, dim_t size,
                        BinaryFunction op, IndexFunction index_a) {
    auto index_it = thrust::make_transform_iterator(thrust::counting_iterator<dim_t>(0), index_a);
    auto a_it = thrust::make_permutation_iterator(a, index_it);
    THRUST_CALL(thrust::transform, a_it, a_it + size, b, c, op);
  }

  // perm_fun is a functor that takes the index in the permuted iterator and
  // return the index in the original iterator.
  template <typename T, typename PermFunction>
  void permute(const T* x, T* y, dim_t size, PermFunction perm_fun) {
    auto ind_it = thrust::counting_iterator<dim_t>(0);
    auto perm_ind_it = thrust::make_transform_iterator(ind_it, perm_fun);
    auto perm_it = thrust::make_permutation_iterator(x, perm_ind_it);
    THRUST_CALL(thrust::copy_n, perm_it, size, y);
  }


  static const cuda::CachingAllocatorConfig allocator_config = cuda::get_caching_allocator_config();
  static hipcub::CachingDeviceAllocator allocator(
    allocator_config.bin_growth,
    allocator_config.min_bin,
    allocator_config.max_bin,
    allocator_config.max_cached_bytes);

  template<>
  void primitives<Device::CUDA>::set_device(int index) {
    CUDA_CHECK(hipSetDevice(index));
  }

  template<>
  int primitives<Device::CUDA>::get_device() {
    int index;
    CUDA_CHECK(hipGetDevice(&index));
    return index;
  }

  template<>
  void* primitives<Device::CUDA>::alloc_data(dim_t size) {
    void* data = nullptr;
    CUDA_CHECK(allocator.DeviceAllocate(&data, size, cuda::get_cuda_stream()));
    return data;
  }

  template<>
  void primitives<Device::CUDA>::free_data(void* data) {
    CUDA_CHECK(allocator.DeviceFree(data));
  }

  template<>
  void primitives<Device::CUDA>::clear_cache() {
    CUDA_CHECK(allocator.FreeAllCached());
  }

  template<>
  template <typename T>
  T primitives<Device::CUDA>::deref(const T* x, dim_t index) {
    T val = T();
    cross_device_primitives<Device::CUDA, Device::CPU>::copy(x + index, &val, 1);
    return val;
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::fill(T* x, T a, dim_t size) {
    THRUST_CALL(thrust::fill_n, x, size, a);
  }
  template<>
  template <typename T>
  void primitives<Device::CUDA>::strided_fill(T* x, T a, dim_t inc_x, dim_t size) {
    auto it = thrust::make_permutation_iterator(
      x, thrust::make_transform_iterator(thrust::counting_iterator<dim_t>(0),
                                         thrust::placeholders::_1 * inc_x));
    THRUST_CALL(thrust::fill_n, it, size, a);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::copy(const T* x, T* y, dim_t size) {
    CUDA_CHECK(hipMemcpyAsync(y, x, size * sizeof (T),
                               hipMemcpyDeviceToDevice, cuda::get_cuda_stream()));
  }

  template<>
  template <typename T>
  T primitives<Device::CUDA>::sum(const T* array, dim_t size) {
    return THRUST_CALL(thrust::reduce, array, array + size);
  }

  template<>
  template <typename T>
  dim_t primitives<Device::CUDA>::max_element(const T* array, dim_t size) {
    const auto* max = THRUST_CALL(thrust::max_element, array, array + size);
    return static_cast<dim_t>(max - array);
  }

  template<>
  template <typename T>
  T primitives<Device::CUDA>::max(const T* array, dim_t size) {
    const auto* max = THRUST_CALL(thrust::max_element, array, array + size);
    return deref(max, 0);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::add(T a, const T* x, T* y, dim_t size) {
    unary_transform(x, y, size, thrust::placeholders::_1 + a);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::add(const T* a, const T* b, T* c, dim_t size) {
    binary_transform(a, b, c, size, thrust::plus<T>());
  }

  template <typename T>
  struct repeat_vec : thrust::unary_function<T, T> {
    T _size;
    repeat_vec(T size)
      : _size(size) {
    }
    __host__ __device__
    T operator()(const T& i) {
      return i % _size;
    }
  };

  template <typename T>
  struct repeat_vec_depth : thrust::unary_function<T, T> {
    T _size;
    repeat_vec_depth(T size)
      : _size(size) {
    }
    __host__ __device__
    T operator()(const T& i) {
      return i / _size;
    }
  };

  template<>
  template <typename T>
  void primitives<Device::CUDA>::add_batch_broadcast(const T* a, const T* b, T* c,
                                                     dim_t a_size, dim_t b_size) {
    binary_transform(a, b, c, b_size, thrust::plus<T>(), repeat_vec<dim_t>(a_size));
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::add_depth_broadcast(const T* a, const T* b, T* c,
                                                     dim_t a_size, dim_t b_size) {
    binary_transform(a, b, c, b_size, thrust::plus<T>(), repeat_vec_depth<dim_t>(b_size / a_size));
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::sub(const T* a, const T* b, T* c, dim_t size) {
    binary_transform(a, b, c, size, thrust::minus<T>());
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::mul(T a, const T* x, T* y, dim_t size) {
    unary_transform(x, y, size, thrust::placeholders::_1 * a);
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::mul(const T* a, const T* b, T* c, dim_t size) {
    binary_transform(a, b, c, size, thrust::multiplies<T>());
  }

  template<>
  template <typename T>
  void primitives<Device::CUDA>::mul_batch_broadcast(const T* a, const T* b, T* c,
                                                     dim_t a_size, dim_t b_size) {
    binary_transform(a, b, c, b_size, thrust::multiplies<T>(), repeat_vec<dim_t>(a_size));
  }

  struct absolute_maximum_func : public thrust::binary_function<float, float, float> {
    __host__ __device__
    float operator()(float a, float b) {
      return fmaxf(fabsf(a), fabsf(b));
    }
  };

  template <typename T>
  struct quantize_func : public thrust::binary_function<float, float, T> {
    __host__ __device__
    T operator()(float scale, float x) {
      return static_cast<T>(x * scale);
    }
  };

  template<>
  void primitives<Device::CUDA>::quantize_batch(const float* x, float* scales, int8_t* qx,
                                                dim_t batch_size, dim_t depth) {
    const dim_t size = batch_size * depth;

    // Assign 1 key per batch.
    auto keys_it = thrust::make_transform_iterator(thrust::counting_iterator<int>(0),
                                                   repeat_vec_depth<int>(depth));

    // scales = 127.0 / reduce_max(abs(x), axis=1)
    THRUST_CALL(thrust::reduce_by_key,
                keys_it, keys_it + size,
                x,
                thrust::make_discard_iterator(),
                thrust::make_transform_output_iterator(
                  scales, static_cast<float>(127) / thrust::placeholders::_1),
                thrust::equal_to<int>(),
                absolute_maximum_func());

    // qx = x * expand_dims(scales, 1)
    binary_transform(scales, x, qx, size,
                     quantize_func<int8_t>(),
                     repeat_vec_depth<dim_t>(depth));
  }

  template <typename T>
  struct dequantize_func : public thrust::binary_function<float, T, float> {
    __device__
    float operator()(float scale, T x) {
      return __fdividef(static_cast<float>(x), scale);
    }
  };

  template<>
  template<>
  void primitives<Device::CUDA>::dequantize_batch(const int8_t* x, const float* scale, float* y,
                                                  dim_t x_size, dim_t scale_size) {
    binary_transform(scale, x, y, x_size,
                     dequantize_func<int8_t>(),
                     repeat_vec_depth<dim_t>(x_size / scale_size));
  }

  struct rescale_func : public thrust::binary_function<int32_t, thrust::tuple<float, float>, float> {
    __device__
    float operator()(int32_t x, const thrust::tuple<float, float>& scales) {
      return __fdividef(__int2float_rn(x), (thrust::get<0>(scales) * thrust::get<1>(scales)));
    }
  };

  template<>
  void primitives<Device::CUDA>::rescale_output(const int32_t* x,
                                                const float* input_scales,
                                                const float* weight_scales,
                                                float* y,
                                                dim_t batch_size,
                                                dim_t depth) {
    const dim_t size = batch_size * depth;

    // y = x / (expand_dims(input_scales, 1) * expand_dims(weight_scales, 0)
    auto input_scales_it = thrust::make_permutation_iterator(
      input_scales,
      thrust::make_transform_iterator(thrust::counting_iterator<int>(0),
                                      repeat_vec_depth<int>(depth)));
    auto weight_scales_it = thrust::make_permutation_iterator(
      weight_scales,
      thrust::make_transform_iterator(thrust::counting_iterator<int>(0),
                                      repeat_vec<int>(depth)));

    auto scales_it = thrust::make_zip_iterator(thrust::make_tuple(input_scales_it, weight_scales_it));
    THRUST_CALL(thrust::transform,
                x, x + size,
                scales_it,
                y,
                rescale_func());
  }

  struct relu_func : public thrust::unary_function<float, float> {
    __host__ __device__
    float operator()(float x) { return fmaxf(x, 0); }
  };

  template<>
  void primitives<Device::CUDA>::relu(const float* x, float* y, dim_t size) {
    unary_transform(x, y, size, relu_func());
  }

  struct gelu_func : public thrust::unary_function<float, float> {
    float _scale;
    gelu_func(float scale)
      : _scale(scale) {
    }
    __host__ __device__
    float operator()(float x) {
      return 0.5f * x * (1.f + tanhf(_scale * (x + 0.044715f * powf(x, 3.f))));
    }
  };

  template<>
  void primitives<Device::CUDA>::gelu(const float* x, float* y, dim_t size) {
    static const float pi = std::acos(-1.f);
    static const float scale = std::sqrt(2.f / pi);
    unary_transform(x, y, size, gelu_func(scale));
  }

  template <typename T>
  struct perm_indices_2d : public thrust::unary_function<T, T> {
    T _rows, _cols;
    perm_indices_2d(T rows, T cols)
      : _rows(rows)
      , _cols(cols) {
    }
    __host__ __device__
    T operator()(const T& i) const {
      const T i0 = i / _rows;
      const T i1 = i % _rows;
      return i1 * _cols + i0;
    }
  };

  template<>
  template <typename T>
  void primitives<Device::CUDA>::transpose_2d(const T* a, const dim_t* dims, T* b) {
    permute(a, b, dims[0] * dims[1], perm_indices_2d<dim_t>(dims[0], dims[1]));
  }

  template <typename T>
  struct perm_indices_3d : public thrust::unary_function<T, T> {
    T _a_ps0, _a_ps1, _a_ps2; // Permuted strides of the original array.
    T _b_d0, _b_d1, _b_d2;    // Dimension of the permutated array.
    T _b_s0, _b_s1, _b_s2;    // Strides of the permutated array.
    perm_indices_3d(const T* dims, const T* perm) {
      const T a_stride[3] = {dims[1] * dims[2], dims[2], 1};
      _a_ps0 = a_stride[perm[0]];
      _a_ps1 = a_stride[perm[1]];
      _a_ps2 = a_stride[perm[2]];
      _b_d0 = dims[perm[0]];
      _b_d1 = dims[perm[1]];
      _b_d2 = dims[perm[2]];
      _b_s0 = _b_d1 * _b_d2;
      _b_s1 = _b_d2;
      _b_s2 = 1;
    }
    __host__ __device__
    T operator()(const T& i) const {
      const T i0 = i / _b_s0;
      const T i1 = i / _b_s1 % _b_d1;
      const T i2 = i % _b_d2;
      return i0 * _a_ps0 + i1 * _a_ps1 + i2 * _a_ps2;
    }
  };

  template<>
  template <typename T>
  void primitives<Device::CUDA>::transpose_3d(const T* a,
                                              const dim_t* dims,
                                              const dim_t* perm,
                                              T* b) {
    permute(a, b, dims[0] * dims[1] * dims[2], perm_indices_3d<dim_t>(dims, perm));
  }

  template <typename T>
  struct perm_indices_4d : public thrust::unary_function<T, T> {
    T _a_ps0, _a_ps1, _a_ps2, _a_ps3; // Permuted strides of the original array.
    T _b_d0, _b_d1, _b_d2, _b_d3;    // Dimension of the permutated array.
    T _b_s0, _b_s1, _b_s2, _b_s3;    // Strides of the permutated array.
    perm_indices_4d(const T* dims, const T* perm) {
      const T a_stride[4] = {dims[1] * dims[2] * dims[3], dims[2] * dims[3], dims[3], 1};
      _a_ps0 = a_stride[perm[0]];
      _a_ps1 = a_stride[perm[1]];
      _a_ps2 = a_stride[perm[2]];
      _a_ps3 = a_stride[perm[3]];
      _b_d0 = dims[perm[0]];
      _b_d1 = dims[perm[1]];
      _b_d2 = dims[perm[2]];
      _b_d3 = dims[perm[3]];
      _b_s0 = _b_d1 * _b_d2 * _b_d3;
      _b_s1 = _b_d2 * _b_d3;
      _b_s2 = _b_d3;
      _b_s3 = 1;
    }
    __host__ __device__
    T operator()(const T& i) const {
      const T i0 = i / _b_s0;
      const T i1 = i / _b_s1 % _b_d1;
      const T i2 = i / _b_s2 % _b_d2;
      const T i3 = i % _b_d3;
      return i0 * _a_ps0 + i1 * _a_ps1 + i2 * _a_ps2 + i3 * _a_ps3;
    }
  };

  template<>
  template <typename T>
  void primitives<Device::CUDA>::transpose_4d(const T* a,
                                              const dim_t* dims,
                                              const dim_t* perm,
                                              T* b) {
    permute(a, b, dims[0] * dims[1] * dims[2] * dims[3], perm_indices_4d<dim_t>(dims, perm));
  }

  template<>
  template<>
  void primitives<Device::CUDA>::gemm(const float* a, const float* b,
                                      bool transpose_a, bool transpose_b,
                                      dim_t m, dim_t n, dim_t k,
                                      float alpha, float beta,
                                      float* c) {
    // Memo: cuBLAS assumes column-major storage.

    const int lda = transpose_a ? m : k;
    const int ldb = transpose_b ? k : n;
    const int ldc = n;

    const hipblasOperation_t transa = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transb = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    CUBLAS_CHECK(hipblasSgemm(cuda::get_cublas_handle(),
                             transb, transa,
                             n, m, k,
                             &alpha,
                             b, ldb,
                             a, lda,
                             &beta,
                             c, ldc));
  }

  template<>
  template<>
  void primitives<Device::CUDA>::gemm(const int8_t* a, const int8_t* b,
                                      bool transpose_a, bool transpose_b,
                                      dim_t m, dim_t n, dim_t k,
                                      float alpha, float beta,
                                      int32_t* c) {
    const int lda = transpose_a ? m : k;
    const int ldb = transpose_b ? k : n;
    const int ldc = n;

    const hipblasOperation_t transa = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transb = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    int32_t alpha_i = alpha;
    int32_t beta_i = beta;

    // cuBLAS assumes column-major storage, so swap a and b accordingly.
    CUBLAS_CHECK(hipblasGemmEx(cuda::get_cublas_handle(),
                              transb, transa,
                              n, m, k,
                              &alpha_i,
                              b, HIP_R_8I, ldb,
                              a, HIP_R_8I, lda,
                              &beta_i,
                              c, HIP_R_32I, ldc,
                              HIP_R_32I,
                              CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }

  template<>
  template<>
  void primitives<Device::CUDA>::gemm_batch(const float* a, const float* b,
                                            bool transpose_a, bool transpose_b,
                                            dim_t batch_size,
                                            dim_t m, dim_t n, dim_t k,
                                            float alpha, float beta,
                                            float* c) {
    // Memo: cuBLAS assumes column-major storage.

    const int lda = transpose_a ? m : k;
    const int ldb = transpose_b ? k : n;
    const int ldc = n;

    const long long int stridea = m * k;
    const long long int strideb = k * n;
    const long long int stridec = m * n;

    const hipblasOperation_t transa = transpose_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    const hipblasOperation_t transb = transpose_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    CUBLAS_CHECK(hipblasSgemmStridedBatched(cuda::get_cublas_handle(),
                                           transb, transa,
                                           n, m, k,
                                           &alpha,
                                           b, ldb, strideb,
                                           a, lda, stridea,
                                           &beta,
                                           c, ldc, stridec,
                                           batch_size));
  }

  struct exp_func : public thrust::unary_function<float, float> {
    __host__ __device__
    float operator()(float x) { return expf(x); }
  };

  template<>
  void primitives<Device::CUDA>::exp(const float* x, float* y, dim_t size) {
    unary_transform(x, y, size, exp_func());
  }

  struct log_func : public thrust::unary_function<float, float> {
    __host__ __device__
    float operator()(float x) { return logf(x); }
  };

  template<>
  void primitives<Device::CUDA>::log(const float* x, float* y, dim_t size) {
    unary_transform(x, y, size, log_func());
  }

  struct pow_func : public thrust::unary_function<float, float> {
    float _power;
    pow_func(float power)
      : _power(power) {
    }
    __host__ __device__
    float operator()(float x) { return powf(x, _power); }
  };

  template<>
  void primitives<Device::CUDA>::pow(const float* x, float* y, float power, dim_t size) {
    unary_transform(x, y, size, pow_func(power));
  }


  template<>
  template <typename T>
  void cross_device_primitives<Device::CPU, Device::CUDA>::copy(const T* x, T* y, dim_t size) {
    CUDA_CHECK(hipMemcpyAsync(y, x, size * sizeof (T), hipMemcpyHostToDevice, cuda::get_cuda_stream()));
  }

  template<>
  template <typename T>
  void cross_device_primitives<Device::CUDA, Device::CPU>::copy(const T* x, T* y, dim_t size) {
    CUDA_CHECK(hipMemcpyAsync(y, x, size * sizeof (T), hipMemcpyDeviceToHost, cuda::get_cuda_stream()));
  }

#define DECLARE_IMPL(T)                                                 \
  template T                                                            \
  primitives<Device::CUDA>::deref(const T* x, dim_t index);             \
  template void                                                         \
  primitives<Device::CUDA>::fill(T* x, T a, dim_t size);                \
  template void                                                         \
  primitives<Device::CUDA>::strided_fill(T* x, T a, dim_t inc_x, dim_t size); \
  template void                                                         \
  primitives<Device::CUDA>::copy<T>(const T* x, T* y, dim_t size);      \
  template T                                                            \
  primitives<Device::CUDA>::sum(const T* array, dim_t size);            \
  template dim_t                                                        \
  primitives<Device::CUDA>::max_element(const T* array, dim_t size);    \
  template T                                                            \
  primitives<Device::CUDA>::max(const T* array, dim_t size);            \
  template void                                                         \
  primitives<Device::CUDA>::add(T a, const T* x, T* y, dim_t size);     \
  template void                                                         \
  primitives<Device::CUDA>::add(const T* a, const T* b, T* c, dim_t size); \
  template void                                                         \
  primitives<Device::CUDA>::add_batch_broadcast(const T* a, const T* b, \
                                                T* c, dim_t a_size, dim_t b_size); \
  template void                                                         \
  primitives<Device::CUDA>::add_depth_broadcast(const T* a, const T* b, \
                                                T* c, dim_t a_size, dim_t b_size); \
  template void                                                         \
  primitives<Device::CUDA>::sub(const T* a, const T* b, T* c, dim_t size); \
  template void                                                         \
  primitives<Device::CUDA>::mul(T a, const T* x, T* y, dim_t size);     \
  template void                                                         \
  primitives<Device::CUDA>::mul(const T* a, const T* b, T* c, dim_t size); \
  template void                                                         \
  primitives<Device::CUDA>::mul_batch_broadcast(const T* a, const T* b, \
                                                T* c, dim_t a_size, dim_t b_size); \
  template void                                                         \
  primitives<Device::CUDA>::transpose_2d(const T* a,                    \
                                         const dim_t* dims,             \
                                         T* b);                         \
  template void                                                         \
  primitives<Device::CUDA>::transpose_3d(const T* a,                    \
                                         const dim_t* dims,             \
                                         const dim_t* perm,             \
                                         T* b);                         \
  template void                                                         \
  primitives<Device::CUDA>::transpose_4d(const T* a,                    \
                                         const dim_t* dims,             \
                                         const dim_t* perm,             \
                                         T* b);                         \
  template void                                                         \
  cross_device_primitives<Device::CPU, Device::CUDA>::copy<T>(const T*, T*, dim_t); \
  template void                                                         \
  cross_device_primitives<Device::CUDA, Device::CPU>::copy<T>(const T*, T*, dim_t);

  DECLARE_ALL_TYPES(DECLARE_IMPL)

}
